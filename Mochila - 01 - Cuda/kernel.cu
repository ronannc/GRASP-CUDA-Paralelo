#include "hip/hip_runtime.h"
/*Bin Packing - two Dimensional
Problema da mochila com duas dimensoes.
Tal problema consiste em escolher o maior numero de itens que comportem em uma mochila (compartimento)

O dataset osado foi o presente na biblioteca ORLIB (OR-Library)

Cada arquivo contem:
Numero de problemas teste
Identificador do Problema
Capacidade do compartimento, numero de itens, numero de itens presentes na melhor solu��o conhecida.
Tamanho dos itens.
*/

#include <hip/hip_runtime.h>
#include ""
#include "objeto.h"
#include "ParallelGRASP.cuh"
#include "EntradaDeDados.h"
#include "Selection_Sort.h"
#include "time.h"

hipError_t parallel_GRASP(int max_iter, int number_of_itens, int bin_capacity, item *size_of_itens, bool *soluctions, int threads, int blocks, int temperatura, int decaimento_temperatura, int tamanho_RCL, int seed);

int main() {
	//semente para gerador de numeros aleatorios
	srand(time(NULL));
	int seed = rand();
	
	//temperatura inicial e o decaimento da temperatura para SA
	int temperatura = 100;
	int decaimento_temperatura = 1;
	
	//tamanha usado para tornar a gera��o da solu��o inicial e busca aleatoria, se 1 fica modo guloso
	int tamanho_RCL = 10;

	//numero de itera��es que o GRASP fara
	int max_iter = 10;
	
	//quantidade de threads e blocos
	int threads = 10;
	int blocks = 10;

	// numero de elementos
	int quantidade_itens = 0;

	// capacidade mochia
	int capacidade_mochila = 0;

	/*recebendo entrada, tamanho de cada item*/
	entrada_dados(quantidade_itens, capacidade_mochila);

	/*Vetor usado para guardar peso e valor e se esta ou nao na mochila*/
	item *itens;
	itens = (item *)malloc(quantidade_itens * sizeof(item));

	if (!itens) {
		printf("Sem memoria disponivel! (itens)\n");
		exit(1);
	}

	entrada_dados_vetor(itens, quantidade_itens);

	//ordenando os dados com rela��o ao valor/peso
	selection_sort(itens, quantidade_itens);

	//for (int i = 0; i < quantidade_itens; i++) {
	//	printf("%d %d\n", itens[i].peso, itens[i].valor);
	//}

	//system("pause");

	printf("====== Bin Packing - Bi Dimensional ======\n");
	printf("\n");
	printf("              IFMG - Formiga            \n");
	printf(" Desenvolvido por: Ronan Nunes Campos   \n");
	printf(" Matricula: 0011919                     \n");
	printf("\n");
	printf(" Dados do problema           \n\n");
	printf(" Numero de Itens: %d                    \n", quantidade_itens);
	printf(" Capacidade Mochila: %d                 \n", capacidade_mochila);
	printf(" Numero de Threads: %d                   \n", threads);
	printf(" Numero de Blocos: %d                    \n", blocks);
	printf(" Numero de Itera��es: %d                \n", max_iter);
	printf(" Geradas %d soluc�es\n", threads * blocks * max_iter);
	printf(" Rodando na GPU                          \n");
	printf("===========================================\n");

	//vetor para guardar o id dos elementos presentes na sulo��o
	bool *soluctions;
	soluctions = (bool *)malloc(quantidade_itens * threads * blocks * sizeof(bool));

	if (!soluctions) {
		printf("Sem memoria disponivel! (soluctions)\n");
		exit(1);
	}
	//iniciando a solu��o com todos os itens fora da mochila == 0
	for (int i = 0; i < quantidade_itens * threads * blocks; i++) {
		soluctions[i] = 0;
	}

	// Rodando GRASP em paralelo.
	clock_t t0, tf;
	double tempo_gasto;
	int max_valor = 0; int valor = 0; int cont = 0; int aux_id = 0; int cont_id = 0;
	t0 = clock();
	hipError_t cudaStatus = parallel_GRASP(max_iter, quantidade_itens, capacidade_mochila, itens, soluctions, threads, blocks, temperatura, decaimento_temperatura, tamanho_RCL, seed);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "parallel_GRASP failed!");
		system("pause");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		system("pause");
		return 1;
	}

	

	for (int i = 0; i < quantidade_itens * threads * blocks; i++) {
		
		if (cont < quantidade_itens) {
			if (soluctions[i] == 1) {
				valor += itens[cont].valor;
			}
			cont++;
		}
		
		if(cont == quantidade_itens){
			
			if (valor > max_valor) {
				max_valor = valor;
				aux_id = cont_id;
			}
			cont_id++;

			cont = 0; valor = 0;
		}
	}
	tf = clock();

	tempo_gasto = ((double)(tf - t0)) / CLOCKS_PER_SEC;
	printf("\n");
	printf("Tempo total gasto: %lf s\n", tempo_gasto);

	printf("===========================================\n\n");
	printf("max valor: %d\n", max_valor);
	for (int i = quantidade_itens * aux_id; i < quantidade_itens * (aux_id + 1); i++) {
		printf("%d ",soluctions[i]);
	}

	printf("\n");
	printf("fim :)\n\n");

	free(itens);
	free(soluctions);
	system("pause");
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t parallel_GRASP(int max_iter, int quantidade_itens, int capacidade_mochila, item *itens, bool *soluctions, int threads, int blocks, int temperatura, int decaimento_temperatura, int tamanho_RCL, int seed) {

	hipError_t cudaStatus;

	// Get device properties
	int cuda_device = 0;

	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, cuda_device);
	int cdpCapable = (properties.major == 3 && properties.minor >= 5) || properties.major >= 4;

	printf("GPU device %s has compute capabilities (SM %d.%d)\n", properties.name, properties.major, properties.minor);

	if (!cdpCapable) {
		printf("this app requires SM 3.5 or higher to use CUDA Dynamic Parallelism.  Exiting...\n");
		system("pause");
		exit(0);
	}

	item *dev_itens;
	cudaStatus = hipMalloc((void**)&dev_itens, quantidade_itens * sizeof(item));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc dev_size_of_itens failed!");
		goto Error;
	}

	bool *dev_soluctions;
	cudaStatus = hipMalloc((void**)&dev_soluctions, quantidade_itens * threads * blocks * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc dev_vectorValor failed!");
		goto Error;
	}

	// Copy output vector from host to device memory.
	cudaStatus = hipMemcpy(dev_itens, itens, quantidade_itens * sizeof(item), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy size_of_itens failed!");
		goto Error;
	}

	// Copy output vector from host to device memory.
	cudaStatus = hipMemcpy(dev_soluctions, soluctions, quantidade_itens  * blocks * threads * sizeof(bool), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy dev_soluctions failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	parallelGRASP << <blocks, threads >> >(max_iter, quantidade_itens, capacidade_mochila, dev_itens, dev_soluctions, temperatura, decaimento_temperatura, tamanho_RCL, seed);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	hipDeviceSynchronize();

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		//goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(soluctions, dev_soluctions, quantidade_itens * blocks * threads * sizeof(bool), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy soluctions failed!, retornou: %d", cudaStatus);
		goto Error;
	}

Error:
	hipFree(dev_soluctions);
	hipFree(dev_itens);

	return cudaStatus;
}
